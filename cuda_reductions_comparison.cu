/*
steps:

1) nvcc cuda_reductions_comparison.cu -o reductions

2) .\reductions

atomic sum reduction correct!
reduction shared mem parallel sweep sum correct!
reduction warp shuffle sum correct!
reduction 2-stage shared memory sum correct!

3)
ncu -o profile reductions

4)
open/launch Nsight Compute and drag profile.ncu-rep from #3 above to it
*/

/*
note that these "bad" approaches are NOT done below:
(1)
Does NOT work (as every thread is trying to do this, potentially at the same time):
*c += a[i];

(2)
Works but NOT parallel:
atomicAdd(&c, a[i]); https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
   do { \
         hipError_t __err = hipGetLastError(); \
         if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
         } \
   } while (0)

//const size_t N = 8ULL*1024ULL*1024ULL;  // data size
//const size_t N = 256*640; // data size
//const size_t N = 163840; // smaller data size to avoid the grid stride loop taking most of the time
//but should still be large enough to allow enough threads saturating the machine (i.e. high occupancy)
//this is now set to the max# threads for V100 - 80 SMs * 2048 max threads per SM
const size_t N = 24*2048; //RTX 4060 has 24 SMs, 24*2048

const int BLOCK_SIZE = 256;  // CUDA maximum is 1024
// naive atomic reduction kernel
__global__ void atomic_red(const float *gdata, float *out){
   size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
   if (idx < N) atomicAdd(out, gdata[idx]);
}

__global__ void reduce_a(float *gdata, float *out){
      __shared__ float sdata[BLOCK_SIZE];
      int tid = threadIdx.x;
      sdata[tid] = 0.0f;
      size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

      while (idx < N) {  // grid stride loop to load data
         sdata[tid] += gdata[idx];
         idx += gridDim.x*blockDim.x;  
      }

      for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
         __syncthreads();
         if (tid < s)  // parallel sweep reduction
            sdata[tid] += sdata[tid + s];
         }
      if (tid == 0) atomicAdd(out, sdata[0]);
   }

__global__ void reduce_ws(float *gdata, float *out){
   __shared__ float sdata[32];
   int tid = threadIdx.x;
   int idx = threadIdx.x+blockDim.x*blockIdx.x;
   float val = 0.0f;
   unsigned mask = 0xFFFFFFFFU;
   int lane = threadIdx.x % warpSize;
   int warpID = threadIdx.x / warpSize;
   while (idx < N) {  // grid stride loop to load 
      val += gdata[idx];
      idx += gridDim.x*blockDim.x;  
   }

   // 1st warp-shuffle reduction
   for (int offset = warpSize/2; offset > 0; offset >>= 1) 
      val += __shfl_down_sync(mask, val, offset);
   if (lane == 0) sdata[warpID] = val;
   __syncthreads(); // put warp results in shared mem

   // hereafter, just warp 0
   if (warpID == 0){
   // reload val from shared mem if warp existed
      val = (tid < blockDim.x/warpSize)?sdata[lane]:0;

 // final warp-shuffle reduction
      for (int offset = warpSize/2; offset > 0; offset >>= 1) 
         val += __shfl_down_sync(mask, val, offset);

      if  (tid == 0) atomicAdd(out, val);
   }
}

__global__ void reduce(float *gdata, float *out, size_t n){
   __shared__ float sdata[BLOCK_SIZE];
   int tid = threadIdx.x;
   sdata[tid] = 0.0f;
   size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

   while (idx < n) {  // grid stride loop to load data
      sdata[tid] += gdata[idx];
      idx += gridDim.x*blockDim.x;  
   }

   for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
      __syncthreads();
      if (tid < s)  // parallel sweep reduction
         sdata[tid] += sdata[tid + s];
   }
   if (tid == 0) out[blockIdx.x] = sdata[0];
}

int main(){

  float *h_A, *h_sum, *d_A, *d_sum, *d_sums;
  h_A = new float[N];  // allocate space for data in host memory
  h_sum = new float;
  for (int i = 0; i < N; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  //const int blocks = 640; //N is 163840 and BLOCK_SIZE is 256 above, 640 is N/BLOCK_SIZE
  const int blocks = N / BLOCK_SIZE;
  hipMalloc(&d_A, N*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sum, sizeof(float));  // allocate device space for sum
  hipMalloc(&d_sums, blocks*sizeof(float));  // allocate device space for partial sums
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  atomic_red<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("atomic reduction kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("atomic reduction kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("atomic sum reduction incorrect!\n"); return -1;}
  printf("atomic sum reduction correct!\n");

  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  reduce_a<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("reduction w/atomic kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction shared mem parallel sweep kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("reduction shared mem parallel sweep sum incorrect!\n"); return -1;}
  printf("reduction w/atomic sum correct!\n");

  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  reduce_ws<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("reduction warp shuffle kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction warp shuffle kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("reduction warp shuffle sum incorrect!\n"); return -1;}
  printf("reduction warp shuffle sum correct!\n");

  hipMemset(d_sums, 0, blocks*sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  reduce<<<blocks, BLOCK_SIZE>>>(d_A, d_sums, N); // reduce stage 1
  cudaCheckErrors("reduction kernel launch failure");
  reduce<<<1, BLOCK_SIZE>>>(d_sums, d_A, blocks); // reduce stage 2
  cudaCheckErrors("reduction kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_A, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction 2-stage shared memory kernel execution failure or hipMemcpy D2H failure");
  if (*h_sum != (float)N) {printf("reduction 2-stage shared memory sum incorrect!\n"); return -1;}
  printf("reduction 2-stage shared memory sum correct!\n");

  return 0;
}
  
