// vectorizedMemFloat_2DThreadTiling here improved on top of vectorizedMem_2DThreadTiling earlier by using float4
// from https://developer.nvidia.com/blog/cuda-pro-tip-increase-performance-with-vectorized-memory-access/ :
/*
We can improve performance of this operation by using the vectorized load and store instructions LD.E.{64,128} and ST.E.{64,128}. These operations also load and store data but do so in 64- or 128-bit widths. Using vectorized loads reduces the total number of instructions, reduces latency, and improves bandwidth utilization.
*/
// as compared to the last kernel  (cuda_sgemm_smem_2DThreadTiling.cu):
// after transposing matrix A when loading from global mem to shared mem, 
// each thread can access row of transposed A and row of B from smem in a vectorized fashion and cache them in the regs
// positive performance impact is allowing loads from As by vectorized SMEM loads
//
// to vectorized loads/stores from/to GMEM, using this to promise the compiler on 128b alignment:
// reinterpret_cast<float4 *>
//
// note that:
// we need to make sure the data alignment is a multiple of 16 bytes (128b) - 2x2 matrics NOT working (*), while 4x4 worked
// maybe, we will have to pad the leading dimension of the matrix A and matrix B
// in the global memory and the shared memory dimensions have to be carefully chosen
// (*) 2 32b floats -> 2 x 4 bytes = 16 bytes .. can ask AI to modify codes below

//Background info:

// C = alpha*(matrix multiplication of A and B) + beta*C
// A is of size M by K, M rows, K columns
// B is of size K by N, K rows, N columns
// then C is M by N
// for now, C is NOT initialized as beta is set to 0.0 below; alpha set to 1.0

/*
steps:

1) nvcc cuda_sgemm_vectorizedMemFloat4_2DThreadTiling.cu -o cuda_sgemm_vectorizedMemFloat4_2DThreadTiling

2) .\cuda_sgemm_vectorizedMemFloat4_2DThreadTiling

3) ncu -o profile cuda_sgemm_vectorizedMemFloat4_2DThreadTiling.exe

4) launch Nsight Compute, and drag profile.ncu-rep obtained from the last step to Nsight Compute
*/

//#include <cstdio>
//#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
//#include <cstdint>
#include <stdint.h>

// this is just for timing measurments
#include <time.h>

// error checking macro
#define cudaCheckErrors(msg) \
   do { \
         hipError_t __err = hipGetLastError(); \
         if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
         } \
   } while (0)

void print_matrix(const int &m, const int &n, const float *A) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f ", A[i * n + j]);
        }
        std::printf("\n");
    }
}

//const int BLOCKSIZE = 32; // CUDA maximum is 1024 total threads per block, 2d block with 32 in each x/y dimension
//const int BLOCKSIZE_X = BLOCKSIZE; //not used for now
//const int BLOCKSIZE_Y = BLOCKSIZE; //not used for now
//const int BLOCKSIZE_K = BLOCKSIZE; //not used for now
//const int BM = 64;
//const int BN = 64;
const int BM = 128;
const int BN = 128;
const int BK = 8;
const int TM = 8;
const int TN = 8;

//const int DSIZE = 2; //small inputs as initial test -> does NOT work here
//const int DSIZE = 4; //this works
//now, trying a larger dataset
//const int DSIZE = 10; //tested and NOT working
//const int DSIZE = 10000;
//and, trying a much larger dataset
const int DSIZE = 1024*16;

/*
__global__ void sgemm_naive(int M, int N, int K, float alpha, const float *A,
                            const float *B, float beta, float *C) {
  // compute position in C that this thread is responsible for
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < M && y < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) {
      tmp += A[x * K + i] * B[i * N + y];
    }
    // C = α*(A@B)+β*C
    C[x * N + y] = alpha * tmp + beta * C[x * N + y];
  }
}
*/
/*
__global__ void sgemm_globalMem(int M, int N, int K, float alpha, const float* A,
    const float* B, float beta, float* C) {
    // compute position in C that this thread is responsible for
    const int y = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = blockIdx.y * blockDim.y + threadIdx.y;

    // `if` condition is necessary for when M or N aren't multiples of 32.
    if (x < M && y < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
            tmp += A[x * K + i] * B[i * N + y];
        }
        // C = α*(A@B)+β*C
        C[x * N + y] = alpha * tmp + beta * C[x * N + y];
    }
}
*/
__global__ void sgemm_vectorizedMemFloat4_2DThreadTiling(int M, int N, int K, float alpha, float* A,
    float* B, float beta, float* C) {

    const int cRow = blockIdx.y;
    const int cCol = blockIdx.x;

    // BN/TN are the number of threads to span a column - 128/8 = 16
    const int threadCol = threadIdx.x % (BN / TN);
    const int threadRow = threadIdx.x / (BN / TN);
    //const int strideA = (BM * BN) / (TM * TN) / BK; // 32
    //const int strideB = (BM * BN) / (TM * TN) / BN; // 2
    const int rowStrideA = 4 * (BM * BN) / (TM * TN) / BK; // 128 -> change is "4 *" due to float4
    const int rowStrideB = 4 * (BM * BN) / (TM * TN) / BN; // 8 -> change is "4 *" due to float4
    int C_rowIdx;
    int C_colIdx;

    // Move blocktile to beginning of A's row and B's column
    A += cRow * BM * K;
    B += cCol * BN;
    C += cRow * BM * N + cCol * BN;

    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

/*
    const int innerColA = threadIdx.x % BK; // 8 columns since BK==8
    const int innerRowA = threadIdx.x / BK; // .. 256/8 = 32 rows -> 32x8 A blocks (*) loaded per threadblock
    const int innerColB = threadIdx.x % BN; // 128 columns since BN==128
    const int innerRowB = threadIdx.x / BN; // .. 256/128 = 2 rows -> 2x128 B blocks (*) loaded per threadblock
    // (*) blocks, NOT block, due to the new loops (see the ones with loadOffset and stride*) below
*/
    const int innerRowA = threadIdx.x / (BK / 4); // "4" due to float4
    const int innerColA = threadIdx.x % (BK / 4); // "4" due to float4
    const int innerRowB = threadIdx.x / (BN / 4); // "4" due to float4
    const int innerColB = threadIdx.x % (BN / 4); // "4" due to float4

    // allocate thread-local cache for results in registerfile
    float threadResults[TM * TN] = {0.0};
    // register caches for As and Bs
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};

    // outer loop over block tiles
    for (int bkIdx = 0; bkIdx < K; bkIdx += BK) {
        // populate the SMEM caches, transposing A while loading from GMEM to SMEM
        for (int offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {
            float4 tmp = reinterpret_cast<float4 *>(&A[innerRowA * K + innerColA * 4])[0];
            As[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
            As[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
            As[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
            As[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;
        }
        for (int offset = 0; offset + rowStrideB <= BK; offset += rowStrideB) {
            reinterpret_cast<float4 *>(&Bs[innerRowB * BN + innerColB * 4])[0] =
                reinterpret_cast<float4 *>(&B[innerRowB * N + innerColB * 4])[0];
        }
        /* //from before:
        //As[innerRowA * BK + innerColA] = A[innerRowA * K + innerColA]; //last 1DThread Tiling kernel
        //Bs[innerRowB * BN + innerColB] = B[innerRowB * N + innerColB]; //last 1DThread Tiling kernel
        // .. with loadOffset and strideA=32, now each thread-block loading:
        // 4 32x8 A blocks -> 128x8 block of A (instead of just 1 32x8 A block)
        for (int loadOffset = 0; loadOffset < BM; loadOffset += strideA) {
            //As[(innerRowA + loadOffset) * BK + innerColA] = // this is before As was transposed
            //    A[(innerRowA + loadOffset) * K + innerColA];
            As[innerColA * BM + (innerRowA + loadOffset)] = // now, As is transposed
                A[(innerRowA + loadOffset) * K + innerColA];
        }
        // .. with loadOffset and strideB=2, now each thread-block loading: 
        // 4 2x128 B blocks -> 8x128 block of B (instead of just 1 2x128 B block)
        for (int loadOffset = 0; loadOffset < BK; loadOffset += strideB) {
            Bs[(innerRowB + loadOffset) * BN + innerColB] =
                B[(innerRowB + loadOffset) * N + innerColB];
        }
        */
        __syncthreads();

        // advance blocktile for next iteration in the outer loop above (A/B not used rest of outer loop)
        A += BK;     // move BK columns to right
        B += BK * N; // move BK rows down

        // calculate per-thread results
        for (int dotIdx = 0; dotIdx < BK; ++dotIdx) {
        /* //last 1DThread Tiling kernel
        // we make the dotproduct loop the outside loop, which facilitates
        // reuse of the Bs entry, which we can cache in a tmp var.
            float tmp = Bs[dotIdx * BN + threadCol];
            for (int resIdx = 0; resIdx < TM; ++resIdx) {
                threadResults[resIdx] +=
                    As[(threadRow * TM + resIdx) * BK + dotIdx] * tmp;
            }
        */
            // load relevant As & Bs entries into registers
            // -> a column of size TM from As, a row of size TN from Bs
            for (int i = 0; i < TM; ++i) {
                //regM[i] = As[(threadRow * TM + i) * BK + dotIdx]; // this is before As was transposed
                regM[i] = As[dotIdx * BM + threadRow * TM + i]; // note + i at the end now
            }
            for (int i = 0; i < TN; ++i) {
                regN[i] = Bs[dotIdx * BN + threadCol * TN + i];
            }
            // perform outer product on register cache, accumulate into threadResults thru the outer loop
            for (int resIdxM = 0; resIdxM < TM; ++resIdxM) {
                for (int resIdxN = 0; resIdxN < TN; ++resIdxN) {
                    threadResults[resIdxM * TN + resIdxN] +=
                        regM[resIdxM] * regN[resIdxN];
                }
            }
        }
        __syncthreads();
    }

    // write out the results
    /*
    for (int resIdx = 0; resIdx < TM; ++resIdx) {
        C_rowIdx = blockIdx.y * BM + threadIdx.x / BN * TM + resIdx;
        C_colIdx = blockIdx.x * BN + threadIdx.x % BN;
        if (C_rowIdx < M && C_colIdx < N) {
            C[(threadRow * TM + resIdx) * N + threadCol] =
                alpha * threadResults[resIdx] +
                beta * C[(threadRow * TM + resIdx) * N + threadCol];
        }
    }
    */
    for (int resIdxM = 0; resIdxM < TM; ++resIdxM) {
        for (int resIdxN = 0; resIdxN < TN; resIdxN += 4) { //incrementing by 4 for float4, writing 4x now 
            C_rowIdx = blockIdx.y * BM + threadRow * TM + resIdxM;
            C_colIdx = blockIdx.x * BN + threadCol * TN + resIdxN;
            if (C_rowIdx < M && C_colIdx < N) {
                /*
                C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN] =
                    alpha * threadResults[resIdxM * TN + resIdxN] +
                    beta * C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN];
                */
                float4 tmp = reinterpret_cast<float4 *>(&C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0];
                // perform GEMM update in reg
                tmp.x = alpha * threadResults[resIdxM * TN + resIdxN] + beta * tmp.x;
                tmp.y = alpha * threadResults[resIdxM * TN + resIdxN + 1] + beta * tmp.y;
                tmp.z = alpha * threadResults[resIdxM * TN + resIdxN + 2] + beta * tmp.z;
                tmp.w = alpha * threadResults[resIdxM * TN + resIdxN + 3] + beta * tmp.w;
                // write back
                reinterpret_cast<float4 *>(&C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0] = tmp;
            }
        }
    }
}

int main(int argc, char *argv[]) {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // these are just for timing
    clock_t t0, t1, t2;
    double t1sum=0.0;
    double t2sum=0.0;
    // start timing
    t0 = clock();

    const int m = DSIZE;
    const int n = DSIZE;
    const int k = DSIZE;

    h_A = new float[m*k]; // matrix A with size m * k
    h_B = new float[k*n];
    h_C = new float[m*n];

    float Aval, Bval;

//small inputs as initial test - tested and matching expected result (way below)
    /*
     *   A = | 1.0 | 3.0 |
     *       | 2.0 | 4.0 |
     *
     *   B = | 5.0 | 7.0 |
     *       | 6.0 | 8.0 |
     */
    //Aval = 1.0;
    //Bval = 5.0;
//small inputs as initial test - end

//more inputs to try next
    Aval = 1.0;
    Bval = 100.0;
//more inputs - end

    for (int x = 0; x < k; x++) {
        for (int y = 0; y < m; y++) {
            h_A[y * k + x] = Aval++;
        }
    }
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < k; y++) {
            h_B[y * n + x] = Bval++;
        }
    }

    std::vector<float> C(m * n);
    const float alpha = 1.0;
    const float beta = 0.0;

    printf("m is %d .. k is %d .. n is %d\n", m, k, n);

    if (m<=10 && k<=10 && n<=10) {
        printf("A\n");
        print_matrix(m, k, h_A);
        printf("=====\n");

        printf("B\n");
        print_matrix(k, n, h_B);
        printf("=====\n");    
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, m*k*sizeof(float));
    hipMalloc(&d_B, k*n*sizeof(float));
    hipMalloc(&d_C, m*n*sizeof(float));
    cudaCheckErrors("hipMalloc failure");
    hipMemcpy(d_A, h_A, m*k*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k*n*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    // compute - launch kernel
    // create as many blocks as necessary to map all of C
    //dim3 gridDim((m + block_size - 1)/block_size, (n + block_size - 1)/block_size, 1);
    //dim3 gridDim((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE, 1);
    dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM, 1);

    // 32 * 32 = 1024 threads per block
    //dim3 blockDim(BLOCKSIZE, BLOCKSIZE, 1);
    // 64 * 64 / 8 = 512 threads per block
    dim3 blockDim(BM*BN/(TM*TN), 1, 1);

    // launch the asynchronous execution of the kernel on the device
    // The function call returns immediately on the host
    sgemm_vectorizedMemFloat4_2DThreadTiling<<<gridDim, blockDim>>>(m, n, k, alpha, d_A, d_B, beta, d_C);

    // Copy results back to host
    hipMemcpy(h_C, d_C, m*n*sizeof(float), hipMemcpyDeviceToHost);

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);

    /*
     *   C = | 23.0 | 31.0 |
     *       | 34.0 | 46.0 |
     */
    if (m<=10 && k<=10 && n<=10) {
        printf("C\n");
        print_matrix(m, n, h_C);
        printf("=====\n");
    }

    printf("First and Last elements of A are: %.2f and %.2f\n", h_A[0], h_A[m * k - 1]);
    printf("First and Last elements of B are: %.2f and %.2f\n", h_B[0], h_B[k * n - 1]);
    printf("First and Last elements of C are: %.2f and %.2f\n", h_C[0], h_C[m * n - 1]);

    /* free resources */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("===== The End =====\n");

    return EXIT_SUCCESS;
}