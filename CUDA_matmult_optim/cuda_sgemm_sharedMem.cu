// sharedMem here as compared to naive sgemm and globalMemCoalescedWithinWarp earlier (*)
// let's see how much slower than cublas (cublas_sgemm_alpha1_beta0.cu) and how much faster than globalMemCoalescedWithinWarp.cu
// C = alpha*(matrix multiplication of A and B) + beta*C
// A is of size M by K, M rows, K columns
// B is of size K by N, K rows, N columns
// then C is M by N
// for now, C is NOT initialized as beta is set to 0.0 below; alpha set to 1.0

// (*)
//basically, the next step here is to use the smaller but faster (lower latency and higher bandwidth than global memory) shared memory (SMEM)
//idea is to have BLOCKSIZE x BLOCKSIZE of the matrices in such shared mem, and have work related to such done as much as possible, before loading up the next block
//e.g. to calculate the 1st 32x32 block of C, the 1st super-row (i.e. k/32 32x32 blocks) of A and the 1st super-column (i.e. n/32 32x32 blocks) of B are needed, but only 1 block from A and 1 block from B can be in the shared mem at a time due to its capacity (*)
// (*) note that parameters/codes here may NOT be tuned (or generalized) for such capacity and occupancy, like for various m/k/n/BLOCKSIZE (could have different dimensions in x/y) values and/or each element data type/size (fixed to float32 here)

/*
steps:

1) nvcc cuda_sgemm_sharedMem.cu -o cuda_sgemm_sharedMem

2) .\cuda_sgemm_sharedMem

...

3) ncu -o profile cuda_sgemm_sharedMem.exe

4) launch Nsight Compute, and drag profile.ncu-rep obtained from the last step to Nsight Compute
*/

//#include <cstdio>
//#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>

// this is just for timing measurments
#include <time.h>

// error checking macro
#define cudaCheckErrors(msg) \
   do { \
         hipError_t __err = hipGetLastError(); \
         if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
         } \
   } while (0)

void print_matrix(const int &m, const int &n, const float *A) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f ", A[i * n + j]);
        }
        std::printf("\n");
    }
}

const int BLOCKSIZE = 32; // CUDA maximum is 1024 total threads per block, 2d block with 32 in each x/y dimension
//const int BLOCKSIZE_X = BLOCKSIZE; //not used for now
//const int BLOCKSIZE_Y = BLOCKSIZE; //not used for now
//const int BLOCKSIZE_K = BLOCKSIZE; //not used for now
//const int DSIZE = 2; //small inputs as initial test - tested and matching expected result (way below)
//now, trying a larger dataset
//const int DSIZE = 10; //tested and verified
//const int DSIZE = 10000;
//and, trying a much larger dataset
const int DSIZE = 1024*16;

/*
__global__ void sgemm_naive(int M, int N, int K, float alpha, const float *A,
                            const float *B, float beta, float *C) {
  // compute position in C that this thread is responsible for
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < M && y < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) {
      tmp += A[x * K + i] * B[i * N + y];
    }
    // C = α*(A@B)+β*C
    C[x * N + y] = alpha * tmp + beta * C[x * N + y];
  }
}
*/
/*
__global__ void sgemm_globalMem(int M, int N, int K, float alpha, const float* A,
    const float* B, float beta, float* C) {
    // compute position in C that this thread is responsible for
    const int y = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = blockIdx.y * blockDim.y + threadIdx.y;

    // `if` condition is necessary for when M or N aren't multiples of 32.
    if (x < M && y < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
            tmp += A[x * K + i] * B[i * N + y];
        }
        // C = α*(A@B)+β*C
        C[x * N + y] = alpha * tmp + beta * C[x * N + y];
    }
}
*/
__global__ void sgemm_sharedMem(int M, int N, int K, float alpha, const float* A,
    const float* B, float beta, float* C) {

    const int y = threadIdx.x;
    const int x = threadIdx.y;
    const int C_rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    const int C_colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // advance pointers to the starting positions
    A += blockIdx.y * BLOCKSIZE * K;
    B += blockIdx.x * BLOCKSIZE;
    C += blockIdx.y * BLOCKSIZE * N + blockIdx.x * BLOCKSIZE;

    // Cache a block (or tile) of A and B in shared memory for faster data reuse
    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];

    float tmp = 0.0;

    for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCKSIZE) {
        // Have each thread load one of the elements in A & B from global memory into shared memory.
        // Make y (=threadIdx.x) the consecutive index to allow global memory access coalescing
        As[x * BLOCKSIZE + y] = A[x * K + y];
        Bs[x * BLOCKSIZE + y] = B[x * N + y];

        // block threads in this block until cache is fully populated
        __syncthreads();

        // advance pointers onto next chunk (i.e. block of BLOCKSIZE * BLOCKSIZE)
        A += BLOCKSIZE;
        B += BLOCKSIZE * N;

        // execute the dotproduct on the currently cached block
        for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
            tmp += As[x * BLOCKSIZE + dotIdx] * Bs[dotIdx * BLOCKSIZE + y];
        }
        // need to sync again at the end, to avoid faster threads
        // fetching the next block into the cache before slower threads are done
        __syncthreads();
    }

    if (C_rowIdx < M && C_colIdx < N)
    {
        C[x * N + y] = alpha * tmp + beta * C[x * N + y];
    }
}

int main(int argc, char *argv[]) {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // these are just for timing
    clock_t t0, t1, t2;
    double t1sum=0.0;
    double t2sum=0.0;
    // start timing
    t0 = clock();

    const int m = DSIZE;
    const int n = DSIZE;
    const int k = DSIZE;

    h_A = new float[m*k]; // matrix A with size m * k
    h_B = new float[k*n];
    h_C = new float[m*n];

    float Aval, Bval;

//small inputs as initial test - tested and matching expected result (way below)
    /*
     *   A = | 1.0 | 3.0 |
     *       | 2.0 | 4.0 |
     *
     *   B = | 5.0 | 7.0 |
     *       | 6.0 | 8.0 |
     */
    //Aval = 1.0;
    //Bval = 5.0;
//small inputs as initial test - end

//more inputs to try next
    Aval = 1.0;
    Bval = 100.0;
//more inputs - end

    for (int x = 0; x < k; x++) {
        for (int y = 0; y < m; y++) {
            h_A[y * k + x] = Aval++;
        }
    }
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < k; y++) {
            h_B[y * n + x] = Bval++;
        }
    }

    std::vector<float> C(m * n);
    const float alpha = 1.0;
    const float beta = 0.0;

    printf("m is %d .. k is %d .. n is %d\n", m, k, n);

    if (m<=10 && k<=10 && n<=10) {
        printf("A\n");
        print_matrix(m, k, h_A);
        printf("=====\n");

        printf("B\n");
        print_matrix(k, n, h_B);
        printf("=====\n");    
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, m*k*sizeof(float));
    hipMalloc(&d_B, k*n*sizeof(float));
    hipMalloc(&d_C, m*n*sizeof(float));
    cudaCheckErrors("hipMalloc failure");
    hipMemcpy(d_A, h_A, m*k*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k*n*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    // compute - launch kernel
    // create as many blocks as necessary to map all of C
    //dim3 gridDim((m + block_size - 1)/block_size, (n + block_size - 1)/block_size, 1);
    dim3 gridDim((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE, 1);

    // 32 * 32 = 1024 thread per block
    dim3 blockDim(BLOCKSIZE, BLOCKSIZE, 1);
    // launch the asynchronous execution of the kernel on the device
    // The function call returns immediately on the host
    sgemm_sharedMem<<<gridDim, blockDim>>>(m, n, k, alpha, d_A, d_B, beta, d_C);

    // Copy results back to host
    hipMemcpy(h_C, d_C, m*n*sizeof(float), hipMemcpyDeviceToHost);

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);

    /*
     *   C = | 23.0 | 31.0 |
     *       | 34.0 | 46.0 |
     */
    if (m<=10 && k<=10 && n<=10) {
        printf("C\n");
        print_matrix(m, n, h_C);
        printf("=====\n");
    }

    printf("First and Last elements of A are: %.2f and %.2f\n", h_A[0], h_A[m * k - 1]);
    printf("First and Last elements of B are: %.2f and %.2f\n", h_B[0], h_B[k * n - 1]);
    printf("First and Last elements of C are: %.2f and %.2f\n", h_C[0], h_C[m * n - 1]);

    /* free resources */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("===== The End =====\n");

    return EXIT_SUCCESS;
}